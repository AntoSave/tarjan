#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include "cuda_graph.h"
#include "preprocess.cu"
#include "array.h"
#include "graph.h"

#define THREADxBLOCK 1024

using namespace std;

scc_set_t *SCCs;

void callback(array_int * scc){
    int scc_id;
    scc_id = array_int_get_min(scc);
    scc_set_add(SCCs,scc_id,scc);
}

int main(int argc, char **argv){
    char path_inputfilename[100];
    char output_filename[100];
    int *bitmask, *d_bitmask;
    int *d_adj_lists, *d_adj_list_indexes;
    bool terminate = false;
    bool *d_terminate;

    if(argc != 3 ){
        printf("Error! Wrong or missing parameters. Please run the program specifing the path of the graph to compute and the name the output file.\n");
        exit(1);
    }
    sscanf(argv[1],"%s",path_inputfilename);
    sscanf(argv[2],"%s",output_filename);

    cuda_graph_t *cuda_graph = cuda_graph_load_from_file(path_inputfilename);
    int n_vertices = cuda_graph->n_vertex;
    int n_bitmask = ((n_vertices-1)/32)+1;
    bitmask = new int[n_bitmask](); //Instantiate an array and initialize it to 0
    //Eseguire il kernel
    int gridsize = ((n_vertices-1)/THREADxBLOCK) + 1; //Numero blocchi su una dimensione della griglia
    dim3 dimGrid(gridsize);
    dim3 dimBlock(THREADxBLOCK);

    hipMalloc(&d_adj_lists, cuda_graph->adj_lists_len * sizeof(int));
    hipMemcpy(d_adj_lists, cuda_graph->adj_lists, cuda_graph->adj_lists_len * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&d_adj_list_indexes, (n_vertices + 1) * sizeof(int));
    hipMemcpy(d_adj_list_indexes, cuda_graph->adj_list_indexes, (n_vertices + 1) * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_bitmask, n_bitmask * sizeof(int));
    hipMemset(d_bitmask, 0, n_bitmask * sizeof(int));

    hipMalloc(&d_terminate, sizeof(bool));
    int iter = 0;
    while(!terminate){
        //printf di C++
        cout << "Called " << iter++ << " times" <<endl;
        terminate = true;
        hipMemset(d_terminate, true, sizeof(bool));
        printf("start del kernel\n");
        DeleteTrivialSCCs<<<dimGrid, dimBlock>>>(d_adj_lists, d_adj_list_indexes, n_vertices, d_bitmask, d_terminate);
        hipDeviceSynchronize();
        hipMemcpy(bitmask, d_bitmask, n_bitmask * sizeof(int), hipMemcpyDeviceToHost);
        /*for(int i = 0; i < n_bitmask; i++){
            printf("%x ", bitmask[i]);
        }
        printf("\n");*/
        //device to host
        hipMemcpy(&terminate, d_terminate, sizeof(bool), hipMemcpyDeviceToHost);
    }
    hipMemcpy(bitmask, d_bitmask, n_bitmask * sizeof(int), hipMemcpyDeviceToHost);
    
    //printf("Bitmask: %x\n", bitmask[0]);
    //First
    //00000000.00000000.00000000.00011000 <-- 31-0
    //00000000.00000000.00000000.00010001 <-- 63-32
    //00000000.00000000.00000000.00000101 <-- 95-64
    //00000000.00000000.00000000.00000101 <-- 127-96
    //Second
    //00000000.00000000.00100001.00011000 <-- 31-0
    //00000000.00000000.00000100.00010011 <-- 63-32
    //00000000.00000000.00000000.00010111 <-- 95-64
    //00000000.00000000.00000000.00000111 <-- 127-96
    //Third
    //00000000.10000000.00100011.00011000 <-- 31-0
    //00000000.00001000.00000100.00010111 <-- 63-32
    //00000000.00000000.00000000.10011111 <-- 95-64
    //00000000.00000000.00000000.00000111 <-- 127-96

    //Lanciare tarjan sequenziale
    //STARTTIME(1);
    graph_t* graph = cuda_graph_to_graph(cuda_graph, bitmask);
    graph_print_debug(graph);
    cuda_graph_free(cuda_graph);
    SCCs = scc_set_init();
    //printf("start graph\n");
    //graph_print_debug(graph);
    //ENDTIME(1,time_init);

    //STARTTIME(2);
    graph_tarjan_foreach(graph, callback);
    //ENDTIME(2,time_tarjan);

    //num = graph_get_num_vertex(graph);
    //STARTTIME(3);
    scc_set_save_to_file(SCCs,output_filename);
    hipFree(d_adj_list_indexes);
    hipFree(d_adj_lists);
    hipFree(d_bitmask);
    hipFree(d_terminate);
    graph_free(graph);
    scc_set_free(SCCs);
    delete[] bitmask;
    //ENDTIME(3,time_destroy);
    //printf("%d,%f,%f,%f,",num,time_init,time_destroy,time_tarjan);

    
    return 0;
}