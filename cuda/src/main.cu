#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include "cuda_graph.h"
#include "preprocess.cu"
#include "array.h"
#include "graph.h"
#include "measurement.h"

#define THREADxBLOCK 1024

using namespace std;

scc_set_t *SCCs;

void callback(array_int * scc){
    int scc_id;
    scc_id = array_int_get_min(scc);
    scc_set_add(SCCs,scc_id,scc);
}

int main(int argc, char **argv){
    char path_inputfilename[100];
    char output_filename[100];
    int *bitmask, *d_bitmask;
    int *d_adj_lists, *d_adj_list_indexes;
    bool terminate = false;
    bool *d_terminate;

    int num;
    double temp=0.0,time_tarjan = 0.0,time_init = 0.0,time_preprocess=0.0,time_graph_conversion=0.0,time_destroy=0.0;

    if(argc != 3 ){
        printf("Error! Wrong or missing parameters. Please run the program specifing the path of the graph to compute and the name the output file.\n");
        exit(1);
    }
    sscanf(argv[1],"%s",path_inputfilename);
    sscanf(argv[2],"%s",output_filename);

    STARTTIME(1);
    SCCs = scc_set_init();

    cuda_graph_t *cuda_graph = cuda_graph_load_from_file(path_inputfilename);
    int n_vertices = cuda_graph->n_vertex;
    int n_bitmask = ((n_vertices-1)/32)+1;
    bitmask = new int[n_bitmask](); //Instantiate an array and initialize it to 0
    //Eseguire il kernel
    int gridsize = ((n_vertices-1)/THREADxBLOCK) + 1; //Numero blocchi su una dimensione della griglia
    dim3 dimGrid(gridsize);
    dim3 dimBlock(THREADxBLOCK);

    hipMalloc(&d_adj_lists, cuda_graph->adj_lists_len * sizeof(int));
    hipMemcpy(d_adj_lists, cuda_graph->adj_lists, cuda_graph->adj_lists_len * sizeof(int), hipMemcpyHostToDevice);
    hipMalloc(&d_adj_list_indexes, (n_vertices + 1) * sizeof(int));
    hipMemcpy(d_adj_list_indexes, cuda_graph->adj_list_indexes, (n_vertices + 1) * sizeof(int), hipMemcpyHostToDevice);

    hipMalloc(&d_bitmask, n_bitmask * sizeof(int));
    hipMemset(d_bitmask, 0, n_bitmask * sizeof(int));

    hipMalloc(&d_terminate, sizeof(bool));
    ENDTIME(1,time_init);
    
    int iter = 0;
    while(!terminate){
        //printf di C++
        //cout << "Called " << iter++ << " times" <<endl;
        terminate = true;
        hipMemset(d_terminate, true, sizeof(bool));
        //printf("start del kernel\n");
        
        STARTTIME(2);
        DeleteTrivialSCCs<<<dimGrid, dimBlock>>>(d_adj_lists, d_adj_list_indexes, n_vertices, d_bitmask, d_terminate);
        hipDeviceSynchronize();
        ENDTIME(2,temp);
        time_preprocess += temp;
    
        //hipMemcpy(bitmask, d_bitmask, n_bitmask * sizeof(int), hipMemcpyDeviceToHost);
        /*for(int i = 0; i < n_bitmask; i++){
            printf("%x ", bitmask[i]);
        }
        printf("\n");*/
        //device to host
        hipMemcpy(&terminate, d_terminate, sizeof(bool), hipMemcpyDeviceToHost);
    }
    hipMemcpy(bitmask, d_bitmask, n_bitmask * sizeof(int), hipMemcpyDeviceToHost);
    
    //printf("Bitmask: %x\n", bitmask[0]);
    //First
    //00000000.00000000.00000000.00011000 <-- 31-0
    //00000000.00000000.00000000.00010001 <-- 63-32
    //00000000.00000000.00000000.00000101 <-- 95-64
    //00000000.00000000.00000000.00000101 <-- 127-96
    //Second
    //00000000.00000000.00100001.00011000 <-- 31-0
    //00000000.00000000.00000100.00010011 <-- 63-32
    //00000000.00000000.00000000.00010111 <-- 95-64
    //00000000.00000000.00000000.00000111 <-- 127-96
    //Third
    //00000000.10000000.00100011.00011000 <-- 31-0
    //00000000.00001000.00000100.00010111 <-- 63-32
    //00000000.00000000.00000000.10011111 <-- 95-64
    //00000000.00000000.00000000.00000111 <-- 127-96

    
    STARTTIME(3);
    graph_t* graph = cuda_graph_to_graph(cuda_graph, bitmask);
    ENDTIME(3,time_graph_conversion);

    
    //graph_print_debug(graph);
    
    //printf("start graph\n");
    //graph_print_debug(graph);
    //Lanciare tarjan sequenziale
    STARTTIME(4);
    graph_tarjan_foreach(graph, callback);
    ENDTIME(4,time_tarjan);

    num = graph_get_num_vertex(graph);
    
    STARTTIME(5);
    scc_set_save_to_file(SCCs,output_filename);
    cuda_graph_free(cuda_graph);
    hipFree(d_adj_list_indexes);
    hipFree(d_adj_lists);
    hipFree(d_bitmask);
    hipFree(d_terminate);
    graph_free(graph);
    scc_set_free(SCCs);
    delete[] bitmask;
    ENDTIME(5,time_destroy);
    printf("%d,%f,%f,%f,%f,%f,",num,time_init,time_destroy,time_preprocess,time_graph_conversion,time_tarjan);

    
    return 0;
}