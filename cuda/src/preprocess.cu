#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include "bitarray.h"

__global__ void MatrixMulKernel(int* adj_lists, int* adj_list_indexes, int *bitmask, int n_vertexes) {
    int vertex_id = blockDim.x * blockIdx.x + threadIdx.x;
    //If thread is not associated with a vertex or the vertex has already been eliminated then do nothing
    if(vertex_id < n_vertexes || test_bit(bitmask, vertex_id)){ 
        return;
    }
    int adj_list_start = adj_list_indexes[vertex_id];
    int adj_list_end = adj_list_indexes[vertex_id+1];
    int elim = false;
    //If adjacency list is empty or has a self-loop delete the vertex
    if(adj_list_start == adj_list_end){
        elim = true;
    }
    if(((adj_list_start + 1) == adj_list_end) && adj_lists[adj_list_start] == vertex_id){
        elim = true;
    }

    if(elim == true){
        //delete
        return;
    }

    //If vertex has no incoming edges delete the vertex
    elim = true;
    for(int i = 0; i < adj_list_indexes[n_vertexes]; i++){
        if(adj_lists[i] == vertex_id){
            elim = false;
            break;
        }
    }

    if(elim == true){
        //delete
    }
}