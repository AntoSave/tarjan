#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdio.h>
#include "bitarray.h"

// kernel uses 10 registers
__global__ void DeleteTrivialSCCs(int* adj_lists, int* adj_list_indexes, int n_vertices, int *bitmask, bool *terminate) {
    int vertex_id = blockDim.x * blockIdx.x + threadIdx.x;
    
    //If thread is not associated with a vertex or the vertex has already been eliminated then do nothing
    //printf("vertex: %d vertex_id: %d n_vertices: %d\n", vertex_id, vertex_id, n_vertices);
    if(vertex_id >= n_vertices){
        return;
    }

    if(test_bit(bitmask, vertex_id) != 0){ 
        //printf("vertex: %d has already been eliminated\n", vertex_id);
        return;
    }
    int adj_list_start = adj_list_indexes[vertex_id];
    int adj_list_end = adj_list_indexes[vertex_id+1];
    int elim = false;
    //printf("vertex: %d adj_list_start: %d adj_list_end: %d\n", vertex_id, adj_list_start, adj_list_end);
    //If adjacency list is empty or has a self-loop delete the vertex
    if(adj_list_start == adj_list_end){
        elim = true;
    }
    if(((adj_list_start + 1) == adj_list_end) && adj_lists[adj_list_start] == vertex_id){
        elim = true;
    }

    if(elim == true){
        //printf("Sto eliminando1\n");
        (*terminate) = false;
        set_bit(bitmask, vertex_id);
        //printf("vertex: %d eliminated1\n", vertex_id);
        return;
    }

    //If vertex has no incoming edges delete the vertex
    elim = true;
    //printf("adj_list_indexes[n_vertices]%d\n",adj_list_indexes[n_vertices]);
    for(int i = 0; i < adj_list_indexes[n_vertices]; i++){
        if(adj_lists[i] == vertex_id){ //! Manca una condizione
            //printf("vertex: %d found\n", vertex_id);
            elim = false;
            break;
        }
    }

    if(elim == true){
        //printf("vertex: %d eliminated2\n", vertex_id);
        (*terminate) = false;
        set_bit(bitmask, vertex_id);
    }
}
